
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cstdio>
#define REAL float
#define TCSIZE 16
#define TCSQ 256
#define BSIZE 32
#define PRINTLIMIT 10
#include "kernel.cuh"

void initmat(REAL *m, int nmats, const int val){
    for(int k=0; k<nmats; ++k){
        int off = k*TCSIZE*TCSIZE;
        for(int i=0; i<TCSIZE; ++i){
            for(int j=0; j<TCSIZE; ++j){
                m[off + i*TCSIZE + j] = (val*(k+1));
            }
        }
    }
}

void printmats(REAL *m, int nmats, const char *msg){
    printf("%s:\n", msg);
    for(int k=0; k<nmats; ++k){
        printf("k=%i\n", k);
        int off = k*TCSIZE*TCSIZE;
        for(int i=0; i<TCSIZE; ++i){
            for(int j=0; j<TCSIZE; ++j){
                printf("%.2f ", m[off + i*TCSIZE + j]);
            }
            printf("\n");
        }
    }
}

int main(int argc, char **argv){
    // params
    if(argc != 3){
        fprintf(stderr, "run as ./prog dev nmats\n");
        exit(EXIT_FAILURE);
    }
    int dev = atoi(argv[1]);
    int nmats = atoi(argv[2]);
    int totaln = nmats*(TCSIZE)*(TCSIZE);
    printf("nmats=%i  dev=%i   TCSIZE=%i  totaln=%i\n", nmats, dev, TCSIZE, totaln);
    
    // set device
    hipSetDevice(dev);

    // mallocs
    REAL *A,  *C;
    REAL *Ad, *Cd;
    half *Adh;

    A = (REAL*)malloc(sizeof(REAL)*totaln);
    C = (REAL*)malloc(sizeof(REAL)*totaln);

    hipMalloc(&Ad, sizeof(REAL)*totaln);
    hipMalloc(&Cd, sizeof(REAL)*totaln);

    hipMalloc(&Adh, sizeof(half)*totaln);

    initmat(A, nmats, 1);
    initmat(C, nmats, 0);

    hipMemcpy(Ad, A, sizeof(REAL)*totaln, hipMemcpyHostToDevice);
    hipMemcpy(Cd, C, sizeof(REAL)*totaln, hipMemcpyHostToDevice);

    convertFp32ToFp16 <<< (totaln + 255)/256, 256 >>> (Adh, Ad, totaln);

    dim3 block, grid;

    block = dim3(TCSIZE, 2, 1);
    grid = dim3(nmats, 1, 1);
    warpReduceSumTC<<<grid, block>>>(Adh, Cd, totaln);
    convertFp32ToFp16 <<< (totaln + 255)/256, 256 >>> (Adh, Cd, totaln);
    warpReduceSumTC<<<grid, block>>>(Adh, Cd, totaln);

    hipDeviceSynchronize();
  
    hipMemcpy(A, Ad, sizeof(REAL)*totaln, hipMemcpyDeviceToHost);
    hipMemcpy(C, Cd, sizeof(REAL)*totaln, hipMemcpyDeviceToHost);

    if(nmats < PRINTLIMIT){
        printmats(A, nmats, "[after] mat A:");
        printmats(C, nmats, "[after] mat C:");
    }

    free(A);
    free(C);

    hipFree(Ad);
    hipFree(Cd);
    hipFree(Adh);

    exit(EXIT_SUCCESS);
}

